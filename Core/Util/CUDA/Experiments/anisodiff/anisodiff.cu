#include "hip/hip_runtime.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "hip/driver_types.h"

#define BLOCK_SIZE 8

__global__ void stampalo(float*,int M, int N);

void convert_double2float( double *input_double, float *output_float,int Ntot);
void convert_float2double( float *input_float, double *output_double,int Ntot);
__global__ void initDelta(float* imInD,int M,int N, float k, float lambda, short type);

void mexFunction( int nlhs, mxArray *plhs[],
   int nrhs, const mxArray *prhs[])
{
	int N,M;
    int iter;
    double k,lambda;
    double* imIn, *diffN,*diffS,*diffW,*diffE, *imOut;
    float* fImIn, *fDiffN, *fDiffS, *fDiffW, *fDiffE, *fImOut;    
    float* imInD, *deltaN,*deltaS,*deltaW,*deltaE;
    int size,sizeDiff;
    short type;


	N = mxGetN(prhs[0]);
    M = mxGetM(prhs[0]);
    
	imIn = (double *)mxGetPr(prhs[0]);    
    
    size = M*N*sizeof(float);
 
    
    fImIn = (float *)malloc(size);
    
	mwSize dims[2];
    dims[0] = M;
    dims[1] = N;        
    
    

    
	convert_double2float(imIn,fImIn,N*M);    
    
    hipMalloc((void **)&imInD,size);
	hipMemcpy(imInD,fImIn,size,hipMemcpyHostToDevice);
    


    iter = (int)mxGetScalar(prhs[1]);  
    k = (double)mxGetScalar(prhs[2]);  
    lambda = (double)mxGetScalar(prhs[3]);   
    type = (short)mxGetScalar(prhs[4]);   
 
    
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid(ceil(N / (float)dimBlock.x), ceil(M / (float)dimBlock.y));    
    
 
    for (int i=0;i<iter;++i) {
        initDelta<<<dimGrid,dimBlock>>>(imInD,M,N,(float)k,(float)lambda,type);
    }


    
        
        
	plhs[0]=mxCreateNumericArray(2,dims,mxDOUBLE_CLASS,mxREAL);
     
    
       
    
    imOut = (double *)mxGetPr(plhs[0]);   
    
    fImOut = (float *)malloc(size);
    hipMemcpy(fImOut,imInD,size,hipMemcpyDeviceToHost);      
 
    convert_float2double(fImOut,imOut,N*M); 
    

//	
}

void convert_double2float( double *input_double, float *output_float,int Ntot)
{
    int i;
    for (i = 0; i < Ntot; i++)
    {
                output_float[i] = (float) input_double[i];
    }
}

void convert_float2double( float *input_float, double *output_double,int Ntot)
{
    int i;
    for (i = 0; i < Ntot; i++)
    {
                output_double[i] = (double) input_float[i];
    }
}



__global__ void initDelta(float* imInD,int M,int N, float k, float lambda, short type) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int index = j + i*M;
        
        float deltaN=0, deltaS=0, deltaW=0, deltaE=0;
        float cN, cS, cW, cE;
        
        int indexN = (j)+(i-1)*(M);
        int indexS = (j)+(i+1)*(M);
        int indexW = (j-1)+(i)*(M);
        int indexE = (j+1)+(i)*(M);        
        
        if (i>1)
            deltaN = imInD[indexN]-imInD[index];
        if (i<N)
            deltaS = imInD[indexS]-imInD[index];    
        if (j>1)
            deltaW = imInD[indexW]-imInD[index];    
        if (j<M)
            deltaE = imInD[indexE]-imInD[index];   
        
        if (type==1) {
            cN = exp(-(pow((deltaN / k),2)));
            cS = exp(-(pow((deltaS / k),2)));
            cW = exp(-(pow((deltaW / k),2)));
            cE = exp(-(pow((deltaE / k),2)));  
        } else {
            cN = 1/(1+pow((deltaN / k),2));
            cS = 1/(1+pow((deltaS / k),2));
            cW = 1/(1+pow((deltaW / k),2));
            cE = 1/(1+pow((deltaE / k),2));        
        }
        
        imInD[index] += lambda*(cN*deltaN + cS*deltaS + cW*deltaW + cE*deltaE);      
        

        __syncthreads();    
}